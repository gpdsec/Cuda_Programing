#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ void hello_cuda()
{
    printf("Hello Cuda \n");
}

int main()
{
    hello_cuda <<<4,5>>> ();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}